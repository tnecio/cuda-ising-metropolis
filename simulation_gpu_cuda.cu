#include "hip/hip_runtime.h"
#include "simulation_gpu_cuda.cuh"

template<typename T>
T *copy_array_to_device(int size, T *from) {
    T *to;
    hipMalloc(&to, size * sizeof(T));
    hipMemcpy(to, from, size * sizeof(T), hipMemcpyHostToDevice);
    return to;
}

template float *copy_array_to_device<float>(int, float *);

template int *copy_array_to_device<int>(int, int *);

template<typename T>
hipError_t copy_array_from_device(int size, T *from, T *to) {
    return hipMemcpy(to, from, size * sizeof(T), hipMemcpyDeviceToHost);
}

template hipError_t copy_array_from_device<int>(int, int *, int *);

template hipError_t copy_array_from_device<float>(int, float *, float *);

CudaPRNG::CudaPRNG(unsigned long long int seed, int size_)
        : size(size_) {
    hiprandStatus cs = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandStatus cs2 = hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hipError_t ce = hipMalloc((void **) &where, size_ * sizeof(float));
}

hiprandStatus CudaPRNG::generate() {
    return hiprandGenerateUniform(gen, where, size);
}

// These function are direct analogues of those in simulation / simulation_cpu

//__device__
//float get_spin_energy(int i, struct GeneralisedModelDeviceMemoryParams dev) {
//    float res = 0;
//    float spin = get_spin(i, dev.spins);
//    res += -dev.magnetic_moment *
//           dev.external_field[i] * spin;
//    for (int j = 0; j < dev.n; j++) {
//        res += dev.interaction[i * dev.n + j] * spin * get_spin(j, dev.spins);
//    }
//    return res;
//}

__device__
float get_spin_energy_simple(int i, struct Simple2DModelDeviceMemoryParams dev) {
    float res = 0;
    float spin = dev.spins[i];
    res += -dev.magnetic_moment *
           dev.external_field * spin;

    int xlen = dev.xlen;
    int ylen = dev.ylen;
    int x = i % xlen;
    int y = i / xlen;

    int left = (x - 1) % xlen + y * xlen;
    int right = (x + 1) % xlen + y * xlen;
    int top = x + (y - 1) % ylen * xlen;
    int down = x + (y + 1) % ylen * xlen;

    res += -dev.interaction * spin * dev.spins[left];
    res += -dev.interaction * spin * dev.spins[right];
    res += -dev.interaction * spin * dev.spins[top];
    res += -dev.interaction * spin * dev.spins[down];

    return res;
}

__device__
float get_spin_flip_prob(float energy, float beta) {
    return exp(-beta * energy);
}
//
//__global__
//void flip_spins_stochastically(struct GeneralisedModelDeviceMemoryParams dev) {
//    int i = blockIdx.x * blockDim.x + threadIdx.x;
//    if (i < dev.n) {
//        float energy = get_spin_energy(i, dev);
//        if (energy < 0 ||
//            dev.prng.where[i] < get_spin_flip_prob(energy, dev.beta)) {
//            dev.out_spins[i] = -dev.spins[i];
//        } else {
//            dev.out_spins[i] = dev.spins[i];
//        }
//    }
//}

__global__
void flip_spins_stochastically_simple(struct Simple2DModelDeviceMemoryParams dev, int offset) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int is_odd_position = (i + (i / dev.xlen * !(dev.xlen % 2)) + offset) % 2;
    if (i < dev.n && is_odd_position) {
        float energy = get_spin_energy_simple(i, dev);
        if (energy < 0 ||
            dev.prng.where[i] < get_spin_flip_prob(energy, dev.beta)) {
            dev.spins[i] = -dev.spins[i];
        }
    }
}

//
//void execute_one_step(struct GeneralisedModelDeviceMemoryParams &dev) {
//    int blocks = (dev.n + 255) / 256;
//    int THREADS_PER_BLOCK = 256;
//    dev.prng.generate();
//    flip_spins_stochastically<<<blocks, THREADS_PER_BLOCK>>>(dev);
//}

void execute_one_step_simple(Simple2DModelDeviceMemoryParams &dev, int offset) {
    int blocks = (dev.n + 255) / 256;
    int THREADS_PER_BLOCK = 256;
    flip_spins_stochastically_simple<<<blocks, THREADS_PER_BLOCK>>>(dev, offset);
}
