#include <iostream>

#include "mre.cuh"

template<typename T>
hipError_t copy_array_from_device(int size, T *from, T *to) {
    return hipMemcpy(to, from, size * sizeof(T), hipMemcpyDeviceToHost);
}

template hipError_t copy_array_from_device<int>(int, int *, int *);
template hipError_t copy_array_from_device<float>(int, float *, float *);

CudaPRNG::CudaPRNG(unsigned long long seed, int size_) {
     hiprandStatus cs = hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandStatus cs2 = hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    hipError_t ce = hipMalloc((void **) &where, size_ * sizeof(float));

}

int test() {
    CudaPRNG cuda_prng(2020ULL, 1024);
    hiprandStatus cs = cuda_prng.generate();

    float arr[1024];
    copy_array_from_device(1024, cuda_prng.where, arr);

    for (int i = 0; i < 10; i++) {
        std::cout << i << ": " << arr[i] << std::endl;
    }

    return 0;
}

hiprandStatus CudaPRNG::generate() {
    return hiprandGenerateUniform(gen, where, size);
}
